#include "hip/hip_runtime.h"
/* This code will generate a fractal image. Uses OpenCV, to compile:
   nvcc CudaFinal.cu `pkg-config --cflags --libs opencv`  */
#include <stdio.h>
#include <stdlib.h>
#include <opencv/highgui.h>
#include "utils/cheader.h"


typedef enum color {BLUE, GREEN, RED} Color;

__global__ void convert_to_hsv(unsigned char *src, float *hsv, int width, int heigth, int step, int channels) {
	float r, g, b;
	float h, s, v;
	int ren,col;

	ren = blockIdx.x;
	col = threadIdx.x;

	r = src[(ren * step) + (col * channels) + RED] / 255.0f;
	g = src[(ren * step) + (col * channels) + GREEN] / 255.0f;
	b = src[(ren * step) + (col * channels) + BLUE] / 255.0f;
	
	float max = fmax(r, fmax(g, b));
	float min = fmin(r, fmin(g, b));
	float diff = max - min;
	
	v = max;
	
	if(v == 0.0f) { // black
		h = s = 0.0f;
	} else {
		s = diff / v;
		if(diff < 0.001f) { // grey
			h = 0.0f;
		} else { // color
			if(max == r) {
				h = 60.0f * (g - b)/diff;
				if(h < 0.0f) { h += 360.0f; }
			} else if(max == g) {
				h = 60.0f * (2 + (b - r)/diff);
			} else {
				h = 60.0f * (4 + (r - g)/diff);
			}
		}		
	}
	// confusion line
	float minh=40.0f;
	float maxh=200.0f;
	float minis = 0;
	float maxs = 100;
	float miniv = 0;
	float maxv = 100;
		
	// if conditionals to check the color blindness line, if the pixel is in this line i change the color to other color base shifting the h	
	if (h > minh && h < maxh && s > minis && s < maxs && v > miniv && v < maxv){
		
		hsv[(ren * step) + (col * channels) + RED] =  (float) (h + 140.0f);
		hsv[(ren * step) + (col * channels) + GREEN] = (float) (s);
		hsv[(ren * step) + (col * channels) + BLUE] = (float) (v);
	} else { // this keep the pixel if it is out of the color blindnessline
		hsv[(ren * step) + (col * channels) + RED] =  (float) (h);
		hsv[(ren * step) + (col * channels) + GREEN] = (float) (s);
		hsv[(ren * step) + (col * channels) + BLUE] = (float) (v);
	}
	
	
}

__global__ void convert_to_rgb(float *hsv, unsigned char *dest, int width, int heigth, int step, int channels) {
	float r, g, b;
	float h, s, v;
	int ren,col;

	ren = blockIdx.x;
	col = threadIdx.x;	
	h = hsv[(ren * step) + (col * channels) + RED];
	s = hsv[(ren * step) + (col * channels) + GREEN];
	v = hsv[(ren * step) + (col * channels) + BLUE];
	
	float f = h/60.0f;
	float hi = floorf(f);
	f = f - hi;
	float p = v * (1 - s);
	float q = v * (1 - s * f);
	float t = v * (1 - s * (1 - f));
	
	if(hi == 0.0f || hi == 6.0f) {
		r = v;
		g = t;
		b = p;
	} else if(hi == 1.0f) {
		r = q;
		g = v;
		b = p;
	} else if(hi == 2.0f) {
		r = p;
		g = v;
		b = t;
	} else if(hi == 3.0f) {
		r = p;
		g = q;
		b = v;
	} else if(hi == 4.0f) {
		r = t;
		g = p;
		b = v;
	} else {
		r = v;
		g = p;
		b = q;
	}

	dest[(ren * step) + (col * channels) + RED] =  (unsigned char) __float2uint_rn(255.0f * r);
	dest[(ren * step) + (col * channels) + GREEN] = (unsigned char) __float2uint_rn(255.0f * g);
	dest[(ren * step) + (col * channels) + BLUE] = (unsigned char) __float2uint_rn(255.0f * b);
}


int main(int argc, char* argv[]) {
	int size, step, size2;
	int i;
	double acum; 
	float *dev_hsv;
	unsigned char *dev_src ,*dev_dest;
	
		
	if (argc != 2) {
		printf("usage: %s source_file\n", argv[0]);
		return -1;
	}
	// creation of the matrixes using OpenCV
	IplImage *src = cvLoadImage(argv[1], CV_LOAD_IMAGE_COLOR);
	IplImage *hsv = cvCreateImage(cvSize(src->width, src->height), IPL_DEPTH_32F, 3);
	IplImage *dest = cvCreateImage(cvSize(src->width, src->height), IPL_DEPTH_8U, 3);
	
	// if the source is incorrect
	if (!src) {
		printf("Could not load image file: %s\n", argv[1]);
		return -1;
	}
	
	// calculate the size for the RGB matrixes and the HSV matrix
	size = src->width * src->height * src->nChannels * sizeof(uchar);
	size2 = src->width * src->height * src->nChannels * sizeof(float);

	// Allocate the memory for the matrixes in the GPU
	hipMalloc((void**) &dev_src, size);
	hipMalloc((void**) &dev_hsv, size2);
	hipMalloc((void**) &dev_dest, size);
	
	// Copy the Matrix from the source image to the GPU
	hipMemcpy(dev_src, src->imageData, size, hipMemcpyHostToDevice);
	
	//initialize the timer
	acum = 0;
	step = src->widthStep / sizeof(uchar);

	printf("Starting...\n");
	for (i = 0; i < N; i++) {
		// Start The timer
		start_timer();
		// Call the kernel to convert from RGB to HSV
		convert_to_hsv<<<src->height, src->width>>>(dev_src, dev_hsv, src->width, src->height, step, src->nChannels);
		// copy the matrix to the Host matrix for hsv
		hipMemcpy(hsv->imageData, dev_hsv, size2, hipMemcpyDeviceToHost);
		// Call the kernel to convert from HSV to RGB
		convert_to_rgb<<<src->height, src->width>>>(dev_hsv, dev_dest, src->width, src->height, step, src->nChannels);
		// copy the computed matrix in RGB to the host to be printed
		hipMemcpy(dest->imageData, dev_dest, size, hipMemcpyDeviceToHost);
		// add the time to the timer
		acum += stop_timer();
	}
	
	// free the memory
	hipFree(dev_dest);
	hipFree(dev_hsv);
	hipFree(dev_src);
	
	printf("avg time = %.5lf ms\n", (acum / N));
	
	cvShowImage("Image (Original)", src);
	cvShowImage("Image (Computed)", dest);
	cvWaitKey(0);
	cvDestroyWindow("Image (Original)");
	cvDestroyWindow("Image (Computed)");

	return 0;
}
